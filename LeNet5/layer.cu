#include "hip/hip_runtime.h"
#include "layer.h"
#include <cstdio>
// Constructor
Layer::Layer(int M, int N, int O)
{
	this->M = M;
	this->N = N;
	this->O = O;

	float h_bias[N];
	float h_weight[N][M];
	hipEvent_t start, stop;
  	hipEventCreate(&start);
  	hipEventCreate(&stop);
	output = NULL;
	preact = NULL;
	bias   = NULL;
	weight = NULL;

	for (int i = 0; i < N; ++i) {
		h_bias[i] = 0.5f - float(rand()) / float(RAND_MAX);
		/*h_bias[i] = 0.0f;*/

		for (int j = 0; j < M; ++j) {
			h_weight[i][j] = 0.5f - float(rand()) / float(RAND_MAX);
			/*h_weight[i][j] = 0.05f;*/
		}
	}

	hipMalloc(&output, sizeof(float) * O);
	hipMalloc(&preact, sizeof(float) * O);

	hipMalloc(&bias, sizeof(float) * N);

	hipMalloc(&weight, sizeof(float) * M * N);

	hipMalloc(&d_output, sizeof(float) * O);
	hipMalloc(&d_preact, sizeof(float) * O);
	hipMalloc(&d_weight, sizeof(float) * M * N);
	hipEventRecord(start,0);
	hipMemcpy(bias, h_bias, sizeof(float) * N, hipMemcpyHostToDevice);

	hipMemcpy(weight, h_weight, sizeof(float) * M * N, hipMemcpyHostToDevice);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
  	float milliseconds;
  	hipEventElapsedTime(&milliseconds, start, stop);
	fprintf(stdout ,"millisecond : %f\n", milliseconds);
	hipEventDestroy(start);
  	hipEventDestroy(stop);
	
}

// Destructor
Layer::~Layer()
{
	hipFree(output);
	hipFree(preact);

	hipFree(bias);

	hipFree(weight);

	hipFree(d_output);
	hipFree(d_preact);
	hipFree(d_weight);
}

// Send data one row from dataset to the GPU
void Layer::setOutput(float *data)
{
	hipMemcpy(output, data, sizeof(float) * O, hipMemcpyHostToDevice);
}

// Reset GPU memory between iterations
void Layer::clear()
{
	hipMemset(output, 0x00, sizeof(float) * O);
	hipMemset(preact, 0x00, sizeof(float) * O);
}

void Layer::bp_clear()
{
	hipMemset(d_output, 0x00, sizeof(float) * O);
	hipMemset(d_preact, 0x00, sizeof(float) * O);
	hipMemset(d_weight, 0x00, sizeof(float) * M * N);
}


__device__ float step_function(float v) //Sigmoid function::Activation Function
{
	return 1 / (1 + exp(-v));
}

__global__ void apply_step_function(float *input, float *output, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] = step_function(input[idx]);
	}
}

__global__ void makeError(float *err, float *output, unsigned int Y, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;// find specific index/thread in GPU
	const int size = blockDim.x * gridDim.x; // the size of all index/thread in GPU

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		err[idx] = ((Y == idx ? 1.0f : 0.0f) - output[idx]);
	}
}

__global__ void apply_grad(float *output, float *grad, const int N)
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		output[idx] += dt * grad[idx];
	}
}

//conv1 28*28 to 24*24*6
__global__ void fp_preact_c1(float input[28][28], float preact[6][24][24], float weight[6][5][5])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 5*5*6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 5);
		const int i2 = ((idx /= 5	) % 5);
		const int i3 = ((idx /= 5	) % 6);
		const int i4 = ((idx /= 6	) % 24);
		const int i5 = ((idx /= 24	) % 24);

		atomicAdd(&preact[i3][i4][i5], weight[i3][i1][i2] * input[i4 + i1][i5 + i2]);
	}
}

__global__ void fp_bias_c1(float preact[6][24][24], float bias[6])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		preact[i1][i2][i3] += bias[i1];
	}
}

//pooling 1 24*24*6 to 12*12*6
__global__ void fp_preact_s1(float input[6][24][24], float preact[6][12][12], float weight[1][2][2])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 2*2*6*12*12;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 2);
		const int i2 = ((idx /= 2	) % 2);
		const int i3 = ((idx /= 2	) % 6);
		const int i4 = ((idx /= 6	) % 12);
		const int i5 = ((idx /= 12	) % 12);

		atomicAdd(&preact[i3][i4][i5], (input[i3][i4*2+i1][i5*2+i2] > preact[i3][i4][i5]) * (input[i3][i4 * 2 + i1][i5 * 2 + i2] - preact[i3][i4][i5]));
	}
}

__global__ void fp_bias_s1(float preact[6][12][12], float bias[1])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*12*12;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 12);
		const int i3 = ((idx /= 12	) % 12);

		preact[i1][i2][i3] += bias[0];
	}
}

//conv2 12*12*6 to 8*8*16
__global__ void fp_preact_c2(float input[6][12][12], float preact[16][8][8], float weight[16][6][5][5])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 5*5*6*16*8*8;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 5);
		const int i2 = ((idx /= 5	) % 5);				
		const int i3 = ((idx /= 5	) % 6);
		const int i6 = ((idx /= 6	) % 16);
		const int i4 = ((idx /= 16	) % 8);
		const int i5 = ((idx /= 8	) % 8);

		atomicAdd(&preact[i6][i4][i5], weight[i6][i3][i1][i2] * input[i3][i4 + i1][i5 + i2]);
	}
}

__global__ void fp_bias_c2(float preact[16][8][8], float bias[16])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*8*8;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 8);
		const int i3 = ((idx /= 8	) % 8);

		preact[i1][i2][i3] += bias[i1];
	}
}

//pooling 2 8*8*16 to 4*4*16
__global__ void fp_preact_s2(float input[16][8][8], float preact[16][4][4], float weight[1][2][2])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 2*2*16*4*4;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 2);
		const int i2 = ((idx /= 2	) % 2);
		const int i3 = ((idx /= 2	) % 16);
		const int i4 = ((idx /= 16	) % 4);
		const int i5 = ((idx /= 4	) % 4);

		atomicAdd(&preact[i3][i4][i5], (input[i3][i4*2+i1][i5*2+i2] > preact[i3][i4][i5]) * (input[i3][i4 * 2 + i1][i5 * 2 + i2] - preact[i3][i4][i5]));
		//atomicAdd(&preact[i3][i4][i5], weight[0][i1][i2] * input[i3][i4 * 2 + i1][i5 * 2 + i2]);
	}
}

__global__ void fp_bias_s2(float preact[16][4][4], float bias[1])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*4*4;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 4);
		const int i3 = ((idx /= 4	) % 4);

		preact[i1][i2][i3] += bias[0];
	}
}

//conv3 4*4*16 to 1*1*120
__global__ void fp_preact_c3(float input[16][4][4], float preact[120], float weight[120][16][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 4*4*16*120*1*1;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 4);
		const int i2 = ((idx /= 4	) % 4);				
		const int i3 = ((idx /= 4	) % 16);
		const int i6 = ((idx /= 16	) % 120);
		atomicAdd(&preact[i6], weight[i6][i3][i1][i2] * input[i3][i1][i2]);
	}
}

__global__ void fp_bias_c3(float preact[120], float bias[120])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 120*1*1;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 120);

		preact[i1] += bias[i1];
	}
}



//full connect 1 120 to 84
__global__ void fp_preact_f1(float input[120], float preact[84], float weight[84][120])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 84*120*1*1;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 84);
		const int i2 = ((idx /= 10	) % 120);

		atomicAdd(&preact[i1], weight[i1][i2] * input[i2]);
	}
}

__global__ void fp_bias_f1(float preact[84], float bias[84])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 84;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		preact[idx] += bias[idx];
	}
}

//full connect 2 84 to 10
__global__ void fp_preact_f2(float input[84], float preact[10], float weight[10][84])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*84*1*1;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 84);

		atomicAdd(&preact[i1], weight[i1][i2] * input[i2]);
	}
}

__global__ void fp_bias_f2(float preact[10], float bias[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		preact[idx] += bias[idx];
	}
}


//back prop start
// output to f2
__global__ void bp_weight_f2(float d_weight[10][84], float d_preact[10], float p_output[84])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*84;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 84);

		d_weight[i1][i2] = d_preact[i1] * p_output[i2];
	}
}

__global__ void bp_bias_f2(float bias[10], float d_preact[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		bias[idx] += dt * d_preact[idx];
	}
}

// output to f1
__global__ void bp_output_f1(float d_output[84], float n_weight[10][84], float nd_preact[10])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 10*84;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 10);
		const int i2 = ((idx /= 10	) % 84);

		atomicAdd(&d_output[i2], n_weight[i1][i2] * nd_preact[i1]);
	}
}

__global__ void bp_preact_f1(float d_preact[84], float d_output[84], float preact[84])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 84;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 84);

		const float o = step_function(preact[i1]);

		d_preact[i1] = d_output[i1] * o * (1 - o);
	}
}



__global__ void bp_weight_f1(float d_weight[84][120], float d_preact[84], float p_output[120])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 84*120;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 84);
		const int i2 = ((idx /= 84	) % 120);

		d_weight[i1][i2] = d_preact[i1] * p_output[i2];
	}
}

__global__ void bp_bias_f1(float bias[84], float d_preact[84])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 84;

	for (int idx = N * pos / size; idx < N * (pos+1) / size; ++idx) {
		bias[idx] += dt * d_preact[idx];
	}
}

// output to c3
__global__ void bp_output_c3(float d_output[120], float n_weight[84][120], float nd_preact[84])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 84*120;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 84);
		const int i2 = ((idx /= 84	) % 120);

		atomicAdd(&d_output[i2], n_weight[i1][i2] * nd_preact[i1]);
	}
}

__global__ void bp_preact_c3(float d_preact[120], float d_output[120], float preact[120])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 120;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 120);

		const float o = step_function(preact[i1]);

		d_preact[i1] = d_output[i1] * o * (1 - o);
	}
}

__global__ void bp_weight_c3(float d_weight[120][16][4][4], float d_preact[120], float p_output[16][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 120*16*4*4;
	const float d = 16.0f*4.0f*4.0f;
	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 120);
		const int i2 = ((idx /= 120	) % 16);
		const int i3 = ((idx /= 16	) % 4);
		const int i4 = ((idx /= 4	) % 4);

		atomicAdd(&d_weight[i1][i2][i3][i4], d_preact[i1] * p_output[i2][i3][i4]/d);
	}
}

__global__ void bp_bias_c3(float bias[120], float d_preact[120])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 120;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 120);
		atomicAdd(&bias[i1], dt * d_preact[i1]);
	}
}

// output to s2
__global__ void bp_output_s2(float d_output[16][4][4], float n_weight[120][16][4][4], float nd_preact[120])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 4*4*16*120;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 4);
		const int i2 = ((idx /= 4	) % 4);
		const int i3 = ((idx /= 4	) % 16);
		const int i4 = ((idx /= 16	) % 120);
		atomicAdd(&d_output[i3][i1][i2], n_weight[i4][i3][i1][i2] * nd_preact[i4]);
	}
}

__global__ void bp_preact_s2(float d_preact[16][4][4], float d_output[16][4][4], float preact[16][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*4*4;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 4);
		const int i3 = ((idx /= 4	) % 4);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_s2(float d_weight[1][2][2], float d_preact[16][4][4], float p_output[16][8][8])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*2*2*16*4*4;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 2);
		const int i3 = ((idx /= 2	) % 2);
		const int i4 = ((idx /= 2	) % 16);
		const int i5 = ((idx /= 16	) % 4);
		const int i6 = ((idx /= 4	) % 4);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i4][i5][i6] * p_output[i4][i5 * 2 + i2][i6 * 2 + i3]);
	}
}

__global__ void bp_bias_s2(float bias[1], float d_preact[16][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*4*4;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 4);
		const int i3 = ((idx /= 4	) % 4);
		atomicAdd(&bias[0], dt * d_preact[i1][i2][i3]/N);
	}
}

// output to c2
__global__ void bp_output_c2(float d_output[16][8][8], float n_weight[16][8][8], float nd_preact[16][4][4], float d_preact[16][4][4])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*2*2*16*4*4;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 2);
		const int i3 = ((idx /= 2	) % 2);
		const int i4 = ((idx /= 2	) % 16);
		const int i5 = ((idx /= 16	) % 4);
		const int i6 = ((idx /= 4	) % 4);

		atomicAdd(&d_output[i4][i5 * 2 + i2][i6 * 2 + i3], (n_weight[i4][i5*2 + i2][i6*2 + i3] == d_preact[i4][i5][i6]) * nd_preact[i4][i5][i6]);
	}
}

__global__ void bp_preact_c2(float d_preact[16][8][8], float d_output[16][8][8], float preact[16][8][8])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16 * 8 * 8;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 8);
		const int i3 = ((idx /= 8	) % 8);

		//const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * 1;
	}
}

__global__ void bp_weight_c2(float d_weight[16][6][5][5], float d_preact[16][8][8], float p_output[6][12][12])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*6*5*5*8*8;
	const float d = pow(8.0f, 2.0f);
	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 6);
		const int i3 = ((idx /= 6	) % 5);
		const int i4 = ((idx /= 5	) % 5);
		const int i5 = ((idx /= 5	) % 8);
		const int i6 = ((idx /= 8	) % 8);

		atomicAdd(&d_weight[i1][i2][i3][i4], d_preact[i1][i5][i6] * p_output[i2][i5+i3][i6+i4]/d);
	}
}

__global__ void bp_bias_c2(float bias[16], float d_preact[16][8][8])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*8*8;
	const float d = pow(8.0f, 2.0f);
	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1 ) % 16);
  		const int i2 = ((idx /= 16 ) % 8);
  		const int i3 = ((idx /= 8 ) % 8);

  		atomicAdd(&bias[i1], dt * d_preact[i1][i2][i3] / d);	
	}
}

//output s1
__global__ void bp_output_s1(float d_output[6][12][12], float n_weight[16][6][5][5], float nd_preact[16][8][8])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 16*6*5*5*8*8;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 16);
		const int i2 = ((idx /= 16	) % 6);
		const int i3 = ((idx /= 6	) % 5);
		const int i4 = ((idx /= 5	) % 5);
		const int i5 = ((idx /= 5	) % 8);
		const int i6 = ((idx /= 8	) % 8);

		atomicAdd(&d_output[i2][i3+i5][i4+i6], n_weight[i1][i2][i3][i4] * nd_preact[i1][i5][i6]);
	}
}

__global__ void bp_preact_s1(float d_preact[6][12][12], float d_output[6][12][12], float preact[6][12][12])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*12*12;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 12);
		const int i3 = ((idx /= 12	) % 12);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_s1(float d_weight[1][2][2], float d_preact[6][12][12], float p_output[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*2*2*6*12*12;
	const float d = pow(6.0f, 3.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 2);
		const int i3 = ((idx /= 2	) % 2);
		const int i4 = ((idx /= 2	) % 6);
		const int i5 = ((idx /= 6	) % 12);
		const int i6 = ((idx /= 12	) % 12);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i4][i5][i6] * p_output[i4][i5 * 2 + i2][i6 * 2 + i3]);
	}
}

__global__ void bp_bias_s1(float bias[1], float d_preact[6][12][12])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*12*12;
	//const float d = pow(6.0f, 3.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 12);
		const int i3 = ((idx /= 12	) % 12);

		atomicAdd(&bias[0], dt * d_preact[i1][i2][i3] / N);
	}
}

//output c1
__global__ void bp_output_c1(float d_output[6][24][24], float n_weight[6][24][24], float nd_preact[6][12][12], float d_preact[6][12][12])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 1*2*2*6*12*12;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 1);
		const int i2 = ((idx /= 1	) % 2);
		const int i3 = ((idx /= 2	) % 2);
		const int i4 = ((idx /= 2	) % 6);
		const int i5 = ((idx /= 6	) % 12);
		const int i6 = ((idx /= 12	) % 12);

		atomicAdd(&d_output[i4][i5 * 2 + i2][i6 * 2 + i3], (n_weight[i4][i5*2 + i2][i6*2 + i3] == d_preact[i4][i5][i6]) * nd_preact[i4][i5][i6]);
		//atomicAdd(&d_output[i4][i5 * 2 + i2][i6 * 2 + i3], n_weight[i1][i2][i3] * nd_preact[i4][i5][i6]);
	}
}

__global__ void bp_preact_c1(float d_preact[6][24][24], float d_output[6][24][24], float preact[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		const float o = step_function(preact[i1][i2][i3]);

		d_preact[i1][i2][i3] = d_output[i1][i2][i3] * o * (1 - o);
	}
}

__global__ void bp_weight_c1(float d_weight[6][5][5], float d_preact[6][24][24], float p_output[28][28])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*5*5*24*24;
	const float d = pow(24.0f, 2.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 5);
		const int i3 = ((idx /= 5	) % 5);
		const int i4 = ((idx /= 5	) % 24);
		const int i5 = ((idx /= 24	) % 24);

		atomicAdd(&d_weight[i1][i2][i3], d_preact[i1][i4][i5] * p_output[i4 + i2][i5 + i3] / d);
	}
}

__global__ void bp_bias_c1(float bias[6], float d_preact[6][24][24])
{
	const int pos = blockIdx.x * blockDim.x + threadIdx.x;
	const int size = blockDim.x * gridDim.x;

	const int N = 6*24*24;
	const float d = pow(24.0f, 2.0f);

	for (int n = N * pos / size; n < N * (pos+1) / size; ++n) {
		int idx = n;
		const int i1 = ((idx /= 1	) % 6);
		const int i2 = ((idx /= 6	) % 24);
		const int i3 = ((idx /= 24	) % 24);

		atomicAdd(&bias[i1], dt * d_preact[i1][i2][i3] / d);
	}
}
